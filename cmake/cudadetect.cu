
#include <stdio.h>


static void geo_handle_cuda_error(hipError_t err, const char* file, int line) {
  if (err != hipSuccess) {
    printf("%s in file: %s at line: %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

#define GEO_HANDLE_CUDA_ERROR( err ) (geo_handle_cuda_error(err, __FILE__, __LINE__))


int main() {
  hipDeviceProp_t dev_properties;
  float minimum_cc = 6.0;
  GEO_HANDLE_CUDA_ERROR(hipGetDeviceProperties(&dev_properties, 0));
  float cc = dev_properties.major + ( dev_properties.minor/10.0f );
  if ( cc < minimum_cc ) { 
    printf("Minimum Compute Capability of %2.1f required: %2.1f found. Not Building CUDA Code.\n",
            min_cc, cc);
    return 1;   //fail
  } else {
    printf("-arch=sm%d%d", dev_properties.major, dev_properties.minor);
    return 0; //cudaSUCCESS
  }

}
